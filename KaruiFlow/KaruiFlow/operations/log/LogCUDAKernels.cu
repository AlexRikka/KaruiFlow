#include "hip/hip_runtime.h"
﻿#pragma once
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include ""
#include "../../core/headers/Kernel.h"


namespace karuiflow {
	template<class T>
	class LogCudaKernel : public Kernel {
	public:
		LogCudaKernel() {};
		void forward(std::vector<Storage*> inputs, Storage* output) override;
		void backward(std::vector<Storage*> inputs, std::vector<bool> requiresGrad,
			Storage* outerGradient, std::vector<Storage*> outputGradients) override;

	private:
		int m_ThreadsPerBlock = 256;
	};
}


template<typename T>
__global__ void cudaForwardLog(T* inputData, T* outputData, size_t nElems) {
	int tId = blockDim.x * blockIdx.x + threadIdx.x;
	if (tId < nElems) {
		outputData[tId] = log(inputData[tId]);
	}
}

template<typename T>
__global__ void cudaBackwardLog(T* inputData, T* outerGradient, T* outputGradient, size_t nElems) {
	int tId = blockDim.x * blockIdx.x + threadIdx.x;
	if (tId < nElems) {
		outputGradient[tId] = ((T)1 / inputData[tId]) * outerGradient[tId]);
	}
}


namespace karuiflow {
	template <class T>
	void LogCudaKernel<T>::forward(std::vector<Storage*> inputs, Storage* output) {
		// Cuda kernels are guarantied to receive Storages that store their data
		// on device (cuda device).
		T* inputData = (T*)inputs[0]->getData();
		T* outputData = (T*)output->getData();
		size_t nElems = inputs[0]->getSize();

		int nBlocks = (nElems + m_ThreadsPerBlock - 1) / m_ThreadsPerBlock;

		cudaForwarLog<T><<<nBlocks, m_ThreadsPerBlock>>>(inputData, outputData, nElems);
	}

	template <class T>
	void LogCudaKernel<T>::backward(std::vector<Storage*> inputs, std::vector<bool> requiresGrad,
		Storage* outerGradient, std::vector<Storage*> outputGradients) {
		T* inputData = (T*)inputs[0]->getData();
		T* _outerGradient = (T*)outerGradient->getData();
		T* outputGradient = (T*)outputGradients[0]->getData();
		size_t nElems = inputs[0]->getSize();

		int nBlocks = (nElems + m_ThreadsPerBlock - 1) / m_ThreadsPerBlock;

		if (requiresGrad[0])
			cudaBackwardLog<T><<<nBlocks, m_ThreadsPerBlock>>>(inputData, _outerGradient, outputGradient, nElems);
	}

}
