#include "hip/hip_runtime.h"
﻿#include "ReluCUDAKernels.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include ""

template<typename T>
__global__ void cudaForwardRelu(T* inputData, T* outputData, size_t nElems) {
	int tId = blockDim.x * blockIdx.x + threadIdx.x;
	if (tId < nElems) {
		outputData[tId] = max(inputData[tId], 0);
	}
}


template<typename T>
__global__ void cudaBackwardRelu(T* inputData, T* outerGradient, T* outputGradient, size_t nElems) {
	int tId = blockDim.x * blockIdx.x + threadIdx.x;
	// Equals 1 if input > 0, otherwise equals 0.
	T isGreaterZero = 0;
	if (tId < nElems) {
		isGreaterZero = (T) ((max(inputData[tId], 0)) != 0);
		outputGradient[tId] = isGreaterZero * outerGradient[tId];
	}
}


namespace karuiflow {
	template <class T>
	void ReluCudaKernel<T>::forward(std::vector<Storage*> inputs, Storage* output) {
		// Cuda kernels are guarantied to receive Storages that store their data
		// on device (cuda device).
		T* inputData = inputs[0]->getData();
		T* outputData = output->getData();
		size_t nElems = inputs[0]->getSize();

		cudaForwardRelu<T><<<nBlock, nThreadsPerBlock>>> (inputData, outputData, nElems);
	}

	template <class T>
	void ReluCudaKernel<T>::backward(std::vector<Storage*> inputs, std::vector<bool> requiresGrad,
		Storage* outerGradient, std::vector<Storage*> outputGradients) {
		T* inputData = inputs[0]->getData();
		T* _outerGradient = outerGradient->getData();
		T* outputGradient = outputGradients[0]->getData();
		size_t nElems = inputs[0]->getSize();

		int nBlocks = (nElems + m_ThreadsPerBlock - 1) / m_ThreadsPerBlock;

		if (requiresGrad[0])
			cudaBackwardRelu<T><<<nBlock, m_ThreadsPerBlock>>>(inputData, _outerGradient, outputGradient, nElems);
	}
}
