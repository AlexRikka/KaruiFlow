#include "hip/hip_runtime.h"
﻿#pragma once
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include ""
#include "../../core/headers/Kernel.h"


namespace karuiflow {
	template<class T>
	class SigmoidCudaKernel : public Kernel {
	public:
		SigmoidCudaKernel() {};
		void forward(std::vector<Storage*> inputs, Storage* output) override;
		void backward(std::vector<Storage*> inputs, std::vector<bool> requiresGrad,
			Storage* outerGradient, std::vector<Storage*> outputGradients) override;

	private:
		int m_ThreadsPerBlock = 256;
	};
}


__device__ float sigmoid(float x) {
	return 1.0f / (1 + exp(-x));
}

template<typename T>
__global__ void cudaForwardSigmoid(T* inputData, T* outputData, size_t nElems) {
	int tId = blockDim.x * blockIdx.x + threadIdx.x;
	if (tId < nElems) {
		outputData[tId] = sigmoid(inputData[tId]);
	}
}

template<typename T>
__global__ void cudaBackwardSigmoid(T* inputData, T* outerGradient, T* outputGradient, size_t nElems) {
	int tId = blockDim.x * blockIdx.x + threadIdx.x;
	if (tId < nElems) {
		outputGradient[tId] = sigmoid(outerGradient[tId])*(1 - sigmoid(outerGradient[tId]))*outerGradient[tId];
	}
}


namespace karuiflow {
	template <class T>
	void SigmoidCudaKernel<T>::forward(std::vector<Storage*> inputs, Storage* output) {
		// Cuda kernels are guarantied to receive Storages that store their data
		// on device (cuda device).
		T* inputData = (T*)inputs[0]->getData();
		T* outputData = (T*)output->getData();
		size_t nElems = inputs[0]->getSize();

		int nBlocks = (nElems + m_ThreadsPerBlock - 1) / m_ThreadsPerBlock;

		cudaForwarSigmoid<T><<<nBlocks, m_ThreadsPerBlock>>>(inputData, outputData, nElems);
	}

	template <class T>
	void SigmoidCudaKernel<T>::backward(std::vector<Storage*> inputs, std::vector<bool> requiresGrad,
		Storage* outerGradient, std::vector<Storage*> outputGradients) {
		T* inputData = (T*)inputs[0]->getData();
		T* _outerGradient = (T*)outerGradient->getData();
		T* outputGradient = (T*)outputGradients[0]->getData();
		size_t nElems = inputs[0]->getSize();

		int nBlocks = (nElems + m_ThreadsPerBlock - 1) / m_ThreadsPerBlock;

		if (requiresGrad[0])
			cudaBackwardSigmoid<T><<<nBlocks, m_ThreadsPerBlock>>>(inputData, _outerGradient, outputGradient, nElems);
	}

}
