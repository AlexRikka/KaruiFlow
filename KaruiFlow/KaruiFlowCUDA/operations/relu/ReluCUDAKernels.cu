#include "hip/hip_runtime.h"
﻿#include "operations/relu/ReluCUDAKernels.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include ""

template<typename T>
__global__ void cudaForwardRelu(T* inputData, T* outputData, size_t nElems) {
	int tId = blockDim.x * blockIdx.x + threadIdx.x;
	if (tId < nElems) {
		outputData[tId] = inputData[tId] * (T)(inputData[tId] > 0);
	}
}


template<typename T>
__global__ void cudaBackwardRelu(T* inputData, T* outerGradient, T* outputGradient, size_t nElems) {
	int tId = blockDim.x * blockIdx.x + threadIdx.x;
	// Equals 1 if input > 0, otherwise equals 0.
	T isGreaterZero = 0;
	if (tId < nElems) {
		isGreaterZero = (T) (inputData[tId] > 0);
		outputGradient[tId] = isGreaterZero * outerGradient[tId];
	}
}


namespace karuiflow {
	template <class T>
	void ReluCudaKernel<T>::forward(std::vector<Storage*> inputs, Storage* output) {
		// Cuda kernels are guarantied to receive Storages that store their data
		// on device (cuda device).
		T* inputData = inputs[0]->getData();
		T* outputData = output->getData();
		size_t nElems = inputs[0]->getSize();

		int nBlocks = (nElems + m_ThreadsPerBlock - 1) / m_ThreadsPerBlock;

		cudaForwardRelu<T><<<nBlocks, nThreadsPerBlock>>> (inputData, outputData, nElems);
	}

	template <class T>
	void ReluCudaKernel<T>::backward(std::vector<Storage*> inputs, std::vector<bool> requiresGrad,
		Storage* outerGradient, std::vector<Storage*> outputGradients) {
		T* inputData = inputs[0]->getData();
		T* _outerGradient = outerGradient->getData();
		T* outputGradient = outputGradients[0]->getData();
		size_t nElems = inputs[0]->getSize();

		int nBlocks = (nElems + m_ThreadsPerBlock - 1) / m_ThreadsPerBlock;

		if (requiresGrad[0])
			cudaBackwardRelu<T><<<nBlocks, m_ThreadsPerBlock>>>(inputData, _outerGradient, outputGradient, nElems);
	}


	void ReluFloatCudaKernel::forward(std::vector<Storage*> inputs, Storage* output) {
		// Cuda kernels are guarantied to receive Storages that store their data
		// on device (cuda device).
		float* inputData = (float*)inputs[0]->getData();
		float* outputData = (float*)output->getData();
		size_t nElems = inputs[0]->getSize();

		int nBlocks = (nElems + m_ThreadsPerBlock - 1) / m_ThreadsPerBlock;

		cudaForwardRelu<float> <<<nBlocks, m_ThreadsPerBlock >> > (inputData, outputData, nElems);
	}

	void ReluFloatCudaKernel::backward(std::vector<Storage*> inputs, std::vector<bool> requiresGrad,
		Storage* outerGradient, std::vector<Storage*> outputGradients) {
		float* inputData = (float*)inputs[0]->getData();
		float* _outerGradient = (float*)outerGradient->getData();
		float* outputGradient = (float*)outputGradients[0]->getData();
		size_t nElems = inputs[0]->getSize();

		int nBlocks = (nElems + m_ThreadsPerBlock - 1) / m_ThreadsPerBlock;

		if (requiresGrad[0])
			cudaBackwardRelu<float> <<<nBlocks, m_ThreadsPerBlock >> > (inputData, _outerGradient, outputGradient, nElems);
	}
}
